#include "hip/hip_runtime.h"
/*
 * This file is part of libnv.
 *
 * Copyright (C) 2012 nagadomi@nurs.or.jp
 * 
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License,
 * or any later version.
 * 
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <cutil_inline.h>
#include "nv_core.h"
#include "nv_ip.h"
#include "nv_cuda.h"
#include "nv_cuda_keypoint.h"

#define NV_KEYPOINT_STAR_R(r) NV_ROUND_INT(r / 6.0f)
#define NV_PI2_INV (1.0f / (NV_PI * 2.0f))

#define NV_KEYPOINT_MIN_POINT_R  6       /* ���o�����ŏ��̔��a(�������l...) */
#define NV_KEYPOINT_SCALE_FACTOR 1.090508f// 2^(1/8) 8�X�e�b�v����2�{�ɂȂ� */
#define NV_KEYPOINT_MIN_R        5.187362f/* �T���J�n�̔��a */
#define NV_KEYPOINT_LEVEL        17       /* �T�����锼�a�̊K�w�� */
#define NV_KEYPOINT_DESC_SCALE   2.0f     /* �L�q�q���v�Z����ۂ̔��a�̃X�P�[���B
											 �ς���Ȋ댯�B */

/* �I���G���e�[�V�����̗ʎq���� */
#define NV_KEYPOINT_ORIENTATION_HIST 64

/* ���z�̃T���v�����B
   �����ق��������ʂ����肷�邪�x���Ȃ�B
   (6 * 3 + 1) : ���x�d��
   (6 * 2 + 1) : ����
   (6 * 1 + 1) : ���x�d��
 */
#define NV_KEYPOINT_HIST_SAMPLE (6 * 3 + 1)  


#define BENCHMARK 0

#define NV_CUDA_KEYPOINT_HIST_DIR 8
#define NV_CUDA_KEYPOINT_DESC_M   9
#define NV_CUDA_KEYPOINT_RADIUS_MAX 50

static __device__ float
nv_cuda_star_integral(const nv_matrix_t *integral,
					  const nv_matrix_t *integral_tilted,
					  int row, int col, int r)
{
	float intl_norm, intl_tilt;

	/* �����`�̈�ӂ̒����̔��� */
	const int side_half = NV_ROUND_INT(r * NV_SQRT2_INV);
	const int cs = col - side_half;
	const int rs = row - side_half;
	const int sh2 = side_half * 2;
	
	/* �����`���̐ϕ� */

	intl_norm = NV_INTEGRAL_V(integral, cs, rs, cs + sh2, rs + sh2);

	/* �����`���̐ϕ� */
	intl_tilt =
		NV_MAT3D_V(integral_tilted, (row - r), col, 0)
		- NV_MAT3D_V(integral_tilted, row, (col - r), 0)
		- NV_MAT3D_V(integral_tilted, row, (col + r), 0)
		+ NV_MAT3D_V(integral_tilted, (row + r), col, 0);

	/* ���v���v�Z����(�덷�ɂ���ĕ��ɂȂ邱�Ƃ�����̂�0.0��) */
	if (intl_norm < 0.0f) {
		intl_norm = 0.0f;
	}
	if (intl_tilt < 0.0f) {
		intl_tilt = 0.0f;
	}
	return intl_norm + intl_tilt;
}

static __global__ void
nv_cuda_keypoint_make_scale_space(
								  nv_matrix_t **memo,
								  const float *area_inv_table,
								  const nv_matrix_t *outer_r,
								  const nv_matrix_t *inner_r,
								  const nv_matrix_t *img_integral,
								  const nv_matrix_t *img_integral_tilted)
{
	int i = blockIdx.y;
	int my_m = blockDim.x * blockIdx.x + threadIdx.x;
	int x = my_m % img_integral->cols;
	int y = my_m / img_integral->cols;

	if (1 <= i && i < NV_KEYPOINT_LEVEL - 1) {
		float f_r = NV_MAT_V(outer_r, 0, i);
		const int star_r = NV_KEYPOINT_STAR_R(f_r);
		const int sy = star_r;
		const int ey = img_integral->rows -1 - star_r;
		const int sx = star_r;
		const int ex = img_integral->cols -1 - star_r;
		const float area_inv = area_inv_table[star_r];
		const int side_half = NV_ROUND_INT(star_r * NV_SQRT2_INV);

		if (sy <= y  && y < ey && sx <= x && x < ex) {
			//float v = nv_cuda_star_integral(img_integral, img_integral_tilted, y, x, star_r);
			float v = NV_INTEGRAL_V(img_integral,
				x - side_half, y - side_half,
				x - side_half + (side_half * 2),
				y - side_half + (side_half * 2))
				+ NV_MAT3D_V(img_integral_tilted, (y - star_r), x, 0)
				- NV_MAT3D_V(img_integral_tilted, y, (x - star_r), 0)
				- NV_MAT3D_V(img_integral_tilted, y, (x + star_r), 0)
				+ NV_MAT3D_V(img_integral_tilted, (y + star_r), x, 0);
			if (v < 0.0f) {
				v = 0.0f;
			}
			NV_MAT_V(memo[i], y, x) = v * area_inv;
		}
	}
}


static __device__ float 
nv_cuda_keypoint_scale_diff(const nv_matrix_t *img_integral, 
					   const nv_matrix_t *img_integral_tilted,
					   const float *area_table,
					   const float *area_inv_table,
					   int y, int x,
					   int outer_r, int inner_r)
{
	float inner = nv_cuda_star_integral(
		img_integral, img_integral_tilted, y, x, inner_r);
	float inner_response = inner * area_inv_table[inner_r];
	float outer = nv_cuda_star_integral(img_integral, img_integral_tilted,
		y, x, outer_r);
	float outer_response = (outer - inner) 
		/ (area_table[outer_r] - area_table[inner_r]);

	return outer_response - inner_response;
}

/* 
 * ��f���Ƃɓ����_�i���j�̃X�P�[���𐄒肷��.
 *
 * 
 */
static __global__ void
nv_cuda_keypoint_scale_search(nv_matrix_t *grid_response, 
							  nv_matrix_t *scale_response,
							  const float *area_table,
							  const float *area_inv_table,
							  const nv_matrix_t *  img_integral,
							  const nv_matrix_t *  img_integral_tilted,
							  const nv_matrix_t *  outer_r,
							  const nv_matrix_t *  inner_r,
							  const int img_rows,
							  const int img_cols
							  )
{
	const int my_m = blockDim.x * blockIdx.x + threadIdx.x;
	const int col = (my_m % (img_cols / 2)) * 2;
	const int row = (my_m / (img_cols / 2)) * 2;
	const int thread_idx = my_m;
	const int offset = NV_ROUND_INT(NV_MAT_V(outer_r, 0, 0) * NV_KEYPOINT_DESC_SCALE) & ~1;
	const int erow = img_rows - offset;
	const int ecol = img_cols - offset;

	/* �e��f�ŃX�P�[����Ԃ̋ɓ_�����o */
	if (offset <= row && row < erow && offset <= col && col < ecol) {
		int level_bound = 0, s;
		const int col_idx = col / 2;
		const int row_idx = row / 2;

		for (s = 0; s < NV_KEYPOINT_LEVEL; s += 2) {
			const int i_r = NV_ROUND_INT(NV_MAT_V(inner_r, 0, s));
			const int o_r = NV_ROUND_INT(NV_MAT_V(outer_r, 0, s));
			const int o_r_offset = NV_ROUND_INT(o_r * NV_KEYPOINT_DESC_SCALE);

			if (row - o_r_offset >= 0
				&& col - o_r_offset >= 0
				&& row + o_r_offset < img_rows
				&& col + o_r_offset < img_cols)
			{
				__syncthreads();
				NV_MAT_V(scale_response, thread_idx, s) = nv_cuda_keypoint_scale_diff(
					img_integral, img_integral_tilted, area_table, area_inv_table,
					row, col, o_r, i_r);
				level_bound = s;
			} else {
				break;
			}
		}
		/* �X�P�[����Ԃ̋ɒl���Ƃ� */
		for (s = 0; s < level_bound - 3; s += 2) {
			const float response_0 = NV_MAT_V(scale_response, thread_idx, s);
			const float response_1 = NV_MAT_V(scale_response, thread_idx, s + 2);
			const float response_2 = NV_MAT_V(scale_response, thread_idx, s + 4);
			if (response_1 > 0.0f && response_0 < response_1 && response_2 < response_1) {
				const float response_1_0 = nv_cuda_keypoint_scale_diff(
					img_integral, img_integral_tilted, area_table, area_inv_table,
					row, col,
					NV_ROUND_INT(NV_MAT_V(outer_r, 0, s + 1)),
					NV_ROUND_INT(NV_MAT_V(inner_r, 0, s + 1)));
				const float response_1_2 = nv_cuda_keypoint_scale_diff(
					img_integral, img_integral_tilted, area_table, area_inv_table,
					row, col,
					NV_ROUND_INT(NV_MAT_V(outer_r, 0, s + 3)),
					NV_ROUND_INT(NV_MAT_V(inner_r, 0, s + 3)));

				__syncthreads();
				if (response_1 < response_1_0) {
					if (response_1_0 > response_1_2) {
						// response_1_0���ő�
						NV_MAT3D_V(grid_response, s + 1, row_idx, col_idx) = response_1_0;
					} else {
						// response_1_2���ő�
						NV_MAT3D_V(grid_response, s + 3, row_idx, col_idx) = response_1_2;
					}
				} else {
					if (response_1 > response_1_2) {
						// response_1���ő�
						NV_MAT3D_V(grid_response, s + 2, row_idx, col_idx) = response_1;
					} else {
						// response_1_2���ő�
						NV_MAT3D_V(grid_response, s + 3, row_idx, col_idx) = response_1_2;
					}
				}
			} else if (response_1 < 0.0f && response_0 > response_1 && response_2 > response_1) {
				const float response_1_0 = nv_cuda_keypoint_scale_diff(
					img_integral, img_integral_tilted, area_table, area_inv_table,
					row, col,
					NV_ROUND_INT(NV_MAT_V(outer_r, 0, s + 1)),
					NV_ROUND_INT(NV_MAT_V(inner_r, 0, s + 1)));
				const float response_1_2 = nv_cuda_keypoint_scale_diff(
					img_integral, img_integral_tilted, area_table, area_inv_table,
					row, col,
					NV_ROUND_INT(NV_MAT_V(outer_r, 0, s + 3)),
					NV_ROUND_INT(NV_MAT_V(inner_r, 0, s + 3)));

				__syncthreads();
				if (response_1 > response_1_0) {
					if (response_1_0 < response_1_2) {
						// response_1_0���ŏ�
						NV_MAT3D_V(grid_response, s + 1, row_idx, col_idx) = response_1_0;
					} else {
						// response_1_2���ŏ�
						NV_MAT3D_V(grid_response, s + 3, row_idx, col_idx) = response_1_2;
					}
				} else {
					if (response_1 < response_1_2) {
						// response_1���ŏ�
						NV_MAT3D_V(grid_response, s + 2, row_idx, col_idx) = response_1;
					} else {
						// response_1_2���ŏ�
						NV_MAT3D_V(grid_response, s + 3, row_idx, col_idx) = response_1_2;
					}
				}
			}
		}
	}
}

static __device__ int
nv_cuda_keypoint_edge_like(const nv_matrix_t *img, int offset, int step, int ky, int kx)
{
#if 0
	float dxs = 0.0f;
	float dys = 0.0f;
	float dxdys = 0.0f;
	int y, x;
	float trace;
	float det;
	
	for (y = ky - offset; y <= ky + offset; y += step) {
		for (x = kx - offset; x <= kx + offset; x += step) {
			float dx = NV_MAT_V(img, y, x - step) - NV_MAT_V(img, y, x + step);
			float dy = NV_MAT_V(img,  y - step, x) - NV_MAT_V(img, y + step, x);
			dxs += dx * dx;
			dys += dy * dy;
			dxdys += dx * dy;
		}
	}
	trace = dxs + dys;
	det = dxs * dys - dxdys * dxdys;
	return trace*trace/det >= NV_KEYPOINT_EDGE_THRESH;
#else
	return 0;
#endif
}

static __global__ void
nv_cuda_keypoint_edge_thresh(const int nkeypoint,
							 const nv_matrix_t *keypoints,
							 const nv_matrix_t *outer_r,
							 nv_matrix_t **memo)
{
	const int i = blockIdx.x;
	const int j = threadIdx.x;
	__shared__ float shm[NV_CUDA_KEYPOINT_RADIUS_MAX * 2 + 1][3];

	if (i < nkeypoint) {
		const int s = (int)NV_MAT_V(keypoints, i, NV_KEYPOINT_LEVEL_IDX);
		const int offset = (int)NV_MAT_V(outer_r, 0, s); // max 50
		const int n = (offset * 2 + 1);
		if (j < n) {
			const nv_matrix_t *img = memo[s];
			const int ky = (int)NV_MAT_V(keypoints, i, NV_KEYPOINT_Y_IDX);
			const int kx = (int)NV_MAT_V(keypoints, i, NV_KEYPOINT_X_IDX);
			const int y = ky - offset + j;
			int x;
			float dxs = 0.0f, dys = 0.0f, dxdys = 0.0f;
			for (x = kx - offset; x <= kx + offset; ++x) {
				float dx = NV_MAT_V(img, y, x - 1) - NV_MAT_V(img, y, x + 1);
				float dy = NV_MAT_V(img, y - 1, x) - NV_MAT_V(img, y + 1, x);
				dxs += dx * dx;
				dys += dy * dy;
				dxdys += dx * dy;
			}
			shm[j][0] = dxs;
			shm[j][1] = dys;
			shm[j][2] = dxdys;
			__syncthreads();
			if (j == 0) {
				int k;
				float trace, det;
				dxs = 0.0f, dys = 0.0f, dxdys = 0.0f;
				for (k = 0; k < n; ++k) {
					dxs += shm[k][0];
					dys += shm[k][1];
					dxdys += shm[k][2];
				}
				trace = dxs + dys;
				det = dxs * dys - dxdys * dxdys;
				if (trace * trace / det >= NV_KEYPOINT_EDGE_THRESH) {
					NV_MAT_V(keypoints, i, NV_KEYPOINT_RESPONSE_IDX) = 0.0f; // clear
				}
			}
		}
	}
}

/* �L�[�|�C���g�̑I�� 
*
* ����_��I����������́A
* 1. �X�P�[����ԂŃt�B���^�������ɒl�ɂȂ��Ă���.
* 2. ������臒l��苭��(��Βl���傫���j�D
* 3. "�����X�P�[��"�̋ߖTnn�̉~���ōő�/�ŏ��̃t�B���^������Ԃ��Ă���.
* �ƂȂ�.
* �v�Z�ʂ������̂Ŕ�I�������͂ł��邾�������K�p����D
*/
static __global__ void
nv_cuda_keypoint_select(nv_matrix_t *keypoints,
						int *nkeypoint,
						int *lock_mem,
						const nv_matrix_t *grid_response,
						const nv_matrix_t *outer_r,
						const int img_rows,
						const int img_cols,
						nv_matrix_t **memo
	)
{
	const int my_m = blockDim.x * blockIdx.x + threadIdx.x;
	const int s = threadIdx.y + 1;
	const int col = (my_m % (img_cols / 2)) * 2;
	const int row = (my_m / (img_cols / 2)) * 2;
	const int offset = NV_ROUND_INT(NV_MAT_V(outer_r, 0, 0) * NV_KEYPOINT_DESC_SCALE) & ~1;
	const int erow = img_rows - offset;
	const int ecol = img_cols - offset;

	if (1 <= s && s < NV_KEYPOINT_LEVEL - 1 && 
		offset <= row && row < erow &&
		offset <= col && col < ecol) 
	{
		const int row_idx = row / 2;
		const int col_idx = col / 2;
		int  sy, sx, ey, ex;
		const float response = NV_MAT3D_V(grid_response, s, row_idx, col_idx);

		if ((fabsf(response) < NV_KEYPOINT_THRESH)) {
			/* ������臒l��菬�����ꍇ�͑I�����Ȃ�. 
			* �ɒl�łȂ�������0�������Ă���̂ł����Œe�����
			*/
			return;
		}

		/* �ߖT ���a * NV_KEYPOINT_NN */
		const int nn = NV_ROUND_INT(NV_MAT_V(outer_r, 0, s) * NV_KEYPOINT_NN);

		sy = row - nn;
		sx = col - nn;
		ey = row + nn + 1;
		ex = col + nn + 1;

		if (sy < 0 
			|| sx < 0
			|| ex >= img_cols
			|| ey >= img_rows)
		{
			/* �ߖT�̋�`�͈͂��摜�ɓ���Ȃ��ꍇ�͑I�����Ȃ�. */
			return;
		}
				
		if (response > 0.0f) {
			/* �ő�l�� */
			int y, x;
			for (y = sy; y < ey; y += 2) {
				const int y_idx = y / 2;
				const int cy = abs(row - y);
				const int cx = NV_ROUND_INT(sqrtf(nn * nn - cy * cy));
				int sx;

				sx = (col - cx);
				ex = (col + cx);

				for (x = sx; x < ex; x += 2) {
					if (response < NV_MAT3D_V(grid_response, s, y_idx, x / 2))
					{
						return;
					}
				}
			}
		} else if (response < 0.0f) {
			/* �ŏ��l�� */
			int y, x;
			for (y = sy; y < ey; y += 2) {
				const int y_idx = y / 2;
				const int cy = abs(row - y);
				const int cx = NV_ROUND_INT(sqrtf(nn * nn - cy * cy));
				int sx;

				sx = (col - cx);
				ex = (col + cx);

				for (x = sx; x < ex; x += 2) {
					if (response > NV_MAT3D_V(grid_response, s, y_idx, x / 2)) 
					{
						return;
					}
				}
			}
		}
		/* �����_�I�� */
		bool needlock = true;
		while (needlock) {
			if (atomicCAS(lock_mem, 0, 1) == 0) {
				/* critical section */
				NV_MAT_V(keypoints, *nkeypoint, NV_KEYPOINT_RESPONSE_IDX) = response;
				NV_MAT_V(keypoints, *nkeypoint, NV_KEYPOINT_Y_IDX) = (float)(row);
				NV_MAT_V(keypoints, *nkeypoint, NV_KEYPOINT_X_IDX) = (float)(col);
				NV_MAT_V(keypoints, *nkeypoint, NV_KEYPOINT_RADIUS_IDX) = NV_MAT_V(outer_r, 0, s);
				NV_MAT_V(keypoints, *nkeypoint, NV_KEYPOINT_LEVEL_IDX) = (float)s;
				++*nkeypoint;

				atomicExch(lock_mem, 0);
				needlock = false;
			}
		}
	}
}

typedef struct {
	int i0;
	float v0;
	int i1;
	float v1;
} nv_cuda_keypoint_histdata_t;

/* ���z�q�X�g�O�����̂��߂̃f�[�^���쐬 */
static inline __device__ void
nv_cuda_keypoint_histdata(nv_cuda_keypoint_histdata_t *histdata,
						  const int hist_n, 
						  const int hist_i,
						  const int yi, const int xi,
						  const int ky, const int kx, 
						  float f_r, float angle,
						  const nv_matrix_t *  memo

	)
{
	const int r = NV_ROUND_INT(f_r);
	const int star_r = NV_KEYPOINT_STAR_R(f_r);
	const int star_r2 = star_r * 2;
	const int star_tilted_r = NV_ROUND_INT((float)star_r * NV_SQRT2_INV);
	const int n = NV_KEYPOINT_HIST_SAMPLE;
	const int sy = (ky - r) + star_r2;
	const int ey = (ky + r) - star_r2;
	const int sx = (kx - r) + star_r2;
	const int ex = (kx + r) - star_r2;
	const float step_scale = (ex - sx) / (float)n;
	const float r2 = (float)((ex - sx) / 2) * ((ex - sx) / 2);
	const int angle45 = NV_ROUND_INT(hist_n / 360.0f * 45.0f);

	/* �����_��r�ߖT������z�q�X�g�O�������쐬����. */
	const float yp = ((float)sy + step_scale * yi);
	int y = NV_ROUND_INT(yp);
	const float yd = ((float)ky - y) * ((float)ky - y);

	if (y >= ey) {
		y = ey - 1;
	}
	const int x = NV_ROUND_INT(((float)sx + step_scale * xi));

	if (x <= ex) {
		const float fdist = yd + ((float)kx - x) * ((float)kx - x);
		if (fdist <= r2) {
			float dx[2], dy[2];
			float magnitude[2], theta[2];
			int bin[2];
			const int dist = NV_ROUND_INT(sqrtf(fdist));
			const float g = ((float)dist / (r + 1));
			const float w = expf(-(g * g) / (2.0f * 0.2f));
			nv_cuda_keypoint_histdata_t *p = &histdata[hist_i];

			/* (x, y)�𒆐S�Ƃ����������d�˂�8�̒��_������z�̕����Ƌ��������߂�.
			* �C���X�g�Ȃǂ͋Ǐ��I�ȕω����������̂�8�_������z�����ߕ��ς���.
			*/
			dx[0] = NV_MAT_V(memo, y, x + star_r) - NV_MAT_V(memo, y, x - star_r);
			dy[0] = NV_MAT_V(memo, y + star_r, x) - NV_MAT_V(memo, y - star_r, x);
			magnitude[0] = sqrtf(dx[0] * dx[0] + dy[0] * dy[0]);
			theta[0] = atan2f(dy[0], dx[0]) + NV_PI;
			theta[0] -= angle;
			if (theta[0] < 0.0f) {
				theta[0] = NV_PI * 2.0f + theta[0];
			}
			bin[0] = NV_ROUND_INT((float)hist_n * theta[0] * NV_PI2_INV);
			if (bin[0] >= hist_n) {
				bin[0] -= hist_n;
			}
			p->i0 = bin[0];
			p->v0 = magnitude[0] * w;

			dx[1] = NV_MAT_V(memo, y + star_tilted_r, x + star_tilted_r)
				- NV_MAT_V(memo, y - star_tilted_r, x - star_tilted_r);
			dy[1] = NV_MAT_V(memo, y + star_tilted_r, x - star_tilted_r)
				- NV_MAT_V(memo, y - star_tilted_r, x + star_tilted_r);
			
			magnitude[1] = sqrtf(dx[1] * dx[1] + dy[1] * dy[1]);
			theta[1] = atan2f(dy[1], dx[1]) + NV_PI;
			theta[1] -= angle;
			if (theta[1] < 0.0f) {
				theta[1] = NV_PI * 2.0f + theta[1];
			}
			bin[1] = NV_ROUND_INT((float)hist_n * theta[1] * NV_PI2_INV) + angle45;
			if (bin[1] >= hist_n) {
				bin[1] -= hist_n;
			}
			p->i1 = bin[1];
			p->v1 = magnitude[1] * w;
		}
	}
}

static __global__ void
nv_cuda_keypoint_orientation_histdata(int nkeypoint,
									   nv_matrix_t *keypoints,
									   nv_matrix_t **memo,
									   nv_cuda_keypoint_histdata_t *hist

	)
{
	const int i = blockIdx.x;
	const int yi = threadIdx.y;
	const int xi = threadIdx.x;
	const int hist_i = (i * NV_KEYPOINT_HIST_SAMPLE * NV_KEYPOINT_HIST_SAMPLE) + yi * NV_KEYPOINT_HIST_SAMPLE + xi;

	if (i < nkeypoint 
		&& xi < NV_KEYPOINT_HIST_SAMPLE 
		&& yi < NV_KEYPOINT_HIST_SAMPLE) 
	{
		hist[hist_i].i0 = -1; // deny flag

		nv_cuda_keypoint_histdata(
			hist, NV_KEYPOINT_ORIENTATION_HIST,
			hist_i,	yi, xi,
			NV_ROUND_INT(NV_MAT_V(keypoints, i, NV_KEYPOINT_Y_IDX)),
			NV_ROUND_INT(NV_MAT_V(keypoints, i, NV_KEYPOINT_X_IDX)),
			NV_MAT_V(keypoints, i, NV_KEYPOINT_RADIUS_IDX), 0.0f,
			memo[(int)NV_MAT_V(keypoints, i, NV_KEYPOINT_LEVEL_IDX)]);
	}
}

static __global__ void
nv_cuda_keypoint_orientation(int nkeypoint,
							 nv_matrix_t *keypoints,
							 nv_cuda_keypoint_histdata_t *histdata
							 )
{
	const int i = blockIdx.x;
	const int k = threadIdx.x;
	__shared__ float temp[NV_KEYPOINT_HIST_SAMPLE][NV_KEYPOINT_ORIENTATION_HIST];

	if (i < nkeypoint && k < NV_KEYPOINT_HIST_SAMPLE) {
		const int hist_i = i * NV_KEYPOINT_HIST_SAMPLE * NV_KEYPOINT_HIST_SAMPLE;
		float *hist = &temp[k][0];
		int j, l;
		const int j_offset_i = hist_i + k * NV_KEYPOINT_HIST_SAMPLE;
#pragma unroll
		for (j = 0; j < NV_KEYPOINT_ORIENTATION_HIST; ++j) {
			hist[j] = 0.0f;
		}
#pragma unroll
		for (j = 0; j < NV_KEYPOINT_HIST_SAMPLE; ++j) {
			const nv_cuda_keypoint_histdata_t *p = &histdata[j_offset_i + j];
			if (p->i0 >= 0) {
				hist[p->i0] += p->v0;
				hist[p->i1] += p->v1;
			}
		}
		__syncthreads();
		if (k == 0) {
			int max_n = -1;
			float max_v = -FLT_MAX;
			for (j = 1; j < NV_KEYPOINT_HIST_SAMPLE; ++j) {
				for (l = 0; l < NV_KEYPOINT_ORIENTATION_HIST; ++l) {
					temp[0][l] += temp[j][l];
				}
			}
#pragma unroll
			for (j = 0; j < NV_KEYPOINT_ORIENTATION_HIST; ++j) {
				float v = temp[0][j];
				if (max_v < v) {
					max_v = v;
					max_n = j;
				}
			}
			NV_MAT_V(keypoints, i, NV_KEYPOINT_ORIENTATION_IDX) = ((2.0f * NV_PI) / (float)NV_KEYPOINT_ORIENTATION_HIST) * max_n - NV_PI;
		}
	}
}



/*
 * �����_�̓����x�N�g�����Z�o����.
 * �����x�N�g����, �����_�𒆐S�Ƃ������adesc_r�̉~���
 * 8�_(PI/4����)���𒆐S�Ƃ���
 * ���adesc_r�̉~���̌��z�q�X�g�O����(8bin)�Ƃ���.
 * �܂�, 8 * 8 = 64�����̓����x�N�g���ƂȂ�.
*/
static __constant__ float g_circle_steps[8] = {
	0.0f,
	NV_PI / 4.0f * 1.0f, NV_PI / 4.0f * 2.0f,
	NV_PI / 4.0f * 3.0f, NV_PI / 4.0f * 4.0f,
	NV_PI / 4.0f * 5.0f, NV_PI / 4.0f * 6.0f,
	NV_PI / 4.0f * 7.0f
};
static __global__ void 
nv_cuda_keypoint_desc_histdata(int nkeypoint,
							   const nv_matrix_t *keypoints,
							   nv_matrix_t **memo,
							   nv_cuda_keypoint_histdata_t *histdata)
{
	const int i = blockIdx.x; // keypoints
	const int k = blockIdx.y; // sub descs
	const int xi = threadIdx.x; // sample
	const int yi = threadIdx.y; // sample
	const int hist_i = (i * NV_KEYPOINT_HIST_SAMPLE * NV_KEYPOINT_HIST_SAMPLE * NV_CUDA_KEYPOINT_DESC_M) 
		+ (k * NV_KEYPOINT_HIST_SAMPLE * NV_KEYPOINT_HIST_SAMPLE) + yi * NV_KEYPOINT_HIST_SAMPLE + xi;

	if (i < nkeypoint) {
		const float desc_r = NV_MAT_V(keypoints, i, NV_KEYPOINT_RADIUS_IDX);
		const float angle = NV_PI + NV_MAT_V(keypoints, i, NV_KEYPOINT_ORIENTATION_IDX);

		histdata[hist_i].i0 = -1;

		if (k == NV_CUDA_KEYPOINT_DESC_M - 1) {
			nv_cuda_keypoint_histdata(
				histdata, NV_CUDA_KEYPOINT_HIST_DIR,
				hist_i,
				yi, xi,
				NV_MAT_V(keypoints, i, NV_KEYPOINT_Y_IDX),
				NV_MAT_V(keypoints, i, NV_KEYPOINT_X_IDX),
				desc_r, angle,
				memo[(int)NV_MAT_V(keypoints, i, NV_KEYPOINT_LEVEL_IDX)]
			);
		} else if (k < NV_CUDA_KEYPOINT_DESC_M) {
			float theta = g_circle_steps[k] + angle;
			if (theta > 2.0f * NV_PI) {
				theta = (theta - 2.0f * NV_PI);
			}
			/* ���z�q�X�g�O���������߂�_�Ɉړ� */
			nv_cuda_keypoint_histdata(
				histdata, NV_CUDA_KEYPOINT_HIST_DIR,
				hist_i,
				yi, xi,
				NV_ROUND_INT(desc_r * sinf(theta) + NV_MAT_V(keypoints, i, NV_KEYPOINT_Y_IDX)),
				NV_ROUND_INT(desc_r * cosf(theta) + NV_MAT_V(keypoints, i, NV_KEYPOINT_X_IDX)),
				desc_r, angle,
				memo[(int)NV_MAT_V(keypoints, i, NV_KEYPOINT_LEVEL_IDX)]
			);
		}
	}
}

static __global__ void 
nv_cuda_keypoint_desc(int nkeypoint,
					  nv_matrix_t *desc, 
					  nv_cuda_keypoint_histdata_t *histdata
					  )

{
	__shared__ float temp[NV_KEYPOINT_HIST_SAMPLE][NV_CUDA_KEYPOINT_HIST_DIR];
	__shared__ float scale;
	const int i = blockIdx.x; // keypoint
	const int k = blockIdx.y; // sub desc
	const int l = threadIdx.x; // hist samples

	if (i < nkeypoint && l < NV_KEYPOINT_HIST_SAMPLE && k < NV_CUDA_KEYPOINT_DESC_M) {
		float *hist = &temp[l][0];
		int j;
		const int hist_i = (i * NV_KEYPOINT_HIST_SAMPLE * NV_KEYPOINT_HIST_SAMPLE * NV_CUDA_KEYPOINT_DESC_M) 
			+ (k * NV_KEYPOINT_HIST_SAMPLE * NV_KEYPOINT_HIST_SAMPLE);
		const int j_offset_i = hist_i + l * NV_KEYPOINT_HIST_SAMPLE;

#pragma unroll
		for (j = 0; j < NV_CUDA_KEYPOINT_HIST_DIR; ++j) {
			hist[j] = 0.0f;
		}
#pragma unroll
		for (j = 0; j < NV_KEYPOINT_HIST_SAMPLE; ++j) {
			const nv_cuda_keypoint_histdata_t *p = &histdata[j_offset_i + j];
			if (0 <= p->i0) {
				hist[p->i0] += p->v0;
				hist[p->i1] += p->v1;
			}
		}
		if (l < NV_CUDA_KEYPOINT_HIST_DIR) {
			__syncthreads();
#pragma unroll
			for (j = 1; j < NV_KEYPOINT_HIST_SAMPLE; ++j) {
				temp[0][l] += temp[j][l];
			}
			__syncthreads();

			// vector_normalize
			if (l == 0) {
				float v = 0.0f;
				float norm;
#pragma unroll
				for (j = 0; j < NV_CUDA_KEYPOINT_HIST_DIR; ++j) {
					v += hist[j] * hist[j];
				}
				norm = sqrtf(v);
				if (norm != 0.0f) {
					scale = 1.0f / norm;
				} else {
					scale = 0.0f;
				}
			}
			__syncthreads();
			NV_MAT_V(desc, i, k * NV_CUDA_KEYPOINT_HIST_DIR + l) = temp[0][l] * scale;
		}
	}
}

/*
 * �X�P�[����Ԃ̃t�B���^�T�C�Y���v�Z����.
 *
 * �O���̔��a�͓����̔��a��2�{�Ƃ���.
 * 
 */
static __host__ void 
nv_cuda_keypoint_radius(nv_matrix_t *outer_r, nv_matrix_t *inner_r)
{
	float cur_r = NV_KEYPOINT_MIN_R;
	int prev_r = NV_KEYPOINT_MIN_R - 1;
	int s;
	int r;

	r = NV_ROUND_INT(NV_KEYPOINT_MIN_R);
	NV_MAT_V(inner_r, 0, 0) = (float)r;
	NV_MAT_V(outer_r, 0, 0) = (r * 2.0f);

	for (s = 1; s < NV_KEYPOINT_LEVEL;) {
		cur_r *= NV_KEYPOINT_SCALE_FACTOR;
		r = NV_ROUND_INT(cur_r);
		if (r - prev_r > 1) {
			prev_r = r;
			NV_MAT_V(inner_r, 0, s) = (float)r;
			NV_MAT_V(outer_r, 0, s) = (r * 2.0f);

			++s;
		}
	}
}

static __host__ int
nv_cuda_keypoint_desc_cmp(const void *p1, const void *p2)
{
	const float *f1 = (const float*) p1;
	const float *f2 = (const float*) p2;
	float ab1;
	float ab2;

	/* ���a�Ń\�[�g */
	if (f1[NV_KEYPOINT_RADIUS_IDX] < f2[NV_KEYPOINT_RADIUS_IDX]) {
		return 1;
	} else if (f1[NV_KEYPOINT_RADIUS_IDX] > f2[NV_KEYPOINT_RADIUS_IDX]) {
		return -1;
	}
	/* �����̐�Βl�Ń\�[�g 
	   TODO: 
	*/
	ab1 = fabsf(f1[NV_KEYPOINT_RESPONSE_IDX]);
	ab2 = fabsf(f2[NV_KEYPOINT_RESPONSE_IDX]);
	if (ab1 < ab2) {
		return 1;
	} else if (ab1 > ab2) {
		return -1;
	}

	return 0;
}


int
nv_cuda_keypoint(nv_matrix_t *keypoints,
				 nv_matrix_t *desc,
				 const nv_matrix_t *img,
				 const int channel)
{
	if (img->rows < 16 || img->cols < 16) {
		return 0;
	}
	int i;
	nv_matrix_t *integral = nv_matrix3d_alloc(1, img->rows + 1, img->cols + 1);
	nv_matrix_t *integral_tilted = nv_matrix3d_alloc(1, img->rows + 1, img->cols + 1);
	const int img_rows = integral->rows - 1;
	const int img_cols = integral->cols - 1;
	int nkeypoint = 0;
	nv_matrix_t **memo = NULL;
	nv_matrix_t *inner_r = nv_matrix_alloc(NV_KEYPOINT_LEVEL, 1);
	nv_matrix_t *outer_r = nv_matrix_alloc(NV_KEYPOINT_LEVEL, 1);
	nv_matrix_t *keypoints_tmp = nv_matrix_alloc(keypoints->n, (img_rows / 2) * (img_cols / 2));
	nv_matrix_t **memo_dev = NULL;
	nv_matrix_t *inner_r_dev = NULL;
	nv_matrix_t *outer_r_dev = NULL;
	nv_matrix_t *integral_dev = NULL;
	nv_matrix_t *integral_tilted_dev = NULL;
	float *area_inv_table_dev, *area_table_dev = NULL;
	nv_matrix_t *grid_response_dev = NULL;
	nv_matrix_t *scale_response_dev = NULL;;
	int *nkeypoint_dev = NULL;
	nv_matrix_t *keypoints_tmp_dev = nv_cuda_matrix_clone(keypoints_tmp);
	nv_matrix_t *keypoints_dev = NULL;
	nv_matrix_t *desc_dev = NULL;
	nv_cuda_keypoint_histdata_t *histdata_dev = NULL;

	int *lock_mem_dev;

	long t = nv_clock();
	int max_r;

	/* */
	NV_ASSERT(desc->n == 72);

	nv_cuda_keypoint_radius(outer_r, inner_r);
	nv_integral(integral, img, channel);
	nv_integral_tilted(integral_tilted, img, channel);
  	t = nv_clock();

	/* ���̓f�[�^�쐬 */ 
	inner_r_dev = nv_cuda_matrix_dup(inner_r);
	outer_r_dev = nv_cuda_matrix_dup(outer_r);
	integral_dev = nv_cuda_matrix_dup(integral);
	integral_tilted_dev = nv_cuda_matrix_dup(integral_tilted);

	scale_response_dev = nv_cuda_matrix_alloc(NV_KEYPOINT_LEVEL, (img_cols / 2) * (img_rows / 2));
	max_r = NV_ROUND_INT(NV_MAT_V(outer_r, 0, NV_KEYPOINT_LEVEL-1)) + 4;

	memo = nv_alloc_type(nv_matrix_t *, NV_KEYPOINT_LEVEL);
	for (i = 0; i < NV_KEYPOINT_LEVEL; ++i) {
		nv_matrix_t *memobuf = nv_cuda_matrix_alloc(integral->cols, integral->rows);
		memo[i] = memobuf;
	}

	CUDA_SAFE_CALL(hipMalloc(&memo_dev, sizeof(nv_matrix_t *) * NV_KEYPOINT_LEVEL));
	CUDA_SAFE_CALL(
		hipMemcpy(memo_dev,
		memo,
		sizeof(nv_matrix_t *) * NV_KEYPOINT_LEVEL, 
		hipMemcpyHostToDevice));

	CUDA_SAFE_CALL(
		hipMalloc(&area_inv_table_dev,
		sizeof(float) * max_r));
	CUDA_SAFE_CALL(
		hipMemcpy(area_inv_table_dev, 
		nv_star_integral_area_inv_static, sizeof(float) * max_r, 
		hipMemcpyHostToDevice));

	CUDA_SAFE_CALL(
		hipMalloc(&area_table_dev,
		sizeof(float) * max_r));
	CUDA_SAFE_CALL(
		hipMemcpy(area_table_dev, 
		nv_star_integral_area_static, sizeof(float) * max_r, 
		hipMemcpyHostToDevice));

	grid_response_dev = nv_cuda_matrix3d_alloc_zero(
		img_cols / 2,
		NV_KEYPOINT_LEVEL,
		img_rows / 2);

	CUDA_SAFE_CALL(hipMalloc(&nkeypoint_dev, sizeof(int)));
	CUDA_SAFE_CALL(hipMemset(nkeypoint_dev, 0, sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc(&lock_mem_dev, sizeof(int)));
	CUDA_SAFE_CALL(hipMemset(lock_mem_dev, 0, sizeof(int)));

	/* detect*/
	{
		int m = (img_cols / 2) * (img_rows / 2);
		dim3 blocks(nv_cuda_block(m));
		dim3 threads(nv_cuda_thread(m));

		nv_cuda_keypoint_scale_search<<<blocks, threads>>>(
			grid_response_dev,
			scale_response_dev,
			area_table_dev,
			area_inv_table_dev,
			integral_dev, integral_tilted_dev, outer_r_dev, inner_r_dev, img_rows, img_cols);
		CUT_CHECK_ERROR("nv_cuda_keypoint_scale_search() failed\n");
	}

#if BENCHMARK
	printf("- detect : %ldms\n", nv_clock() - t);
	t = nv_clock();
#endif

	/* run make_scale_space kernel */
	{
		int m = integral->cols * integral->rows;
		dim3 blocks(nv_cuda_block(m), NV_KEYPOINT_LEVEL);
		dim3 threads(nv_cuda_thread(m));
	
		nv_cuda_keypoint_make_scale_space<<<blocks, threads>>>(
			memo_dev,
			area_inv_table_dev,
			outer_r_dev,
			inner_r_dev,
			integral_dev,
			integral_tilted_dev);
		CUT_CHECK_ERROR("nv_cuda_keypoint_make_scale_space() failed\n");

#if BENCHMARK
		printf("- make_scale_space : %ldms\n", nv_clock() - t);
		t = nv_clock();
#endif
	}
	/* select */
	{
		int s = NV_KEYPOINT_LEVEL - 2; // s += 1; 16 <= s
		int m = (img_cols / 2) * (img_rows / 2);
		dim3 select_blocks(m / 8 + (m % 8 == 0 ? 0:1));
		dim3 select_threads(8, s);
		float *tmp;

		nv_cuda_keypoint_select<<<select_blocks, select_threads>>>(
			keypoints_tmp_dev,
			nkeypoint_dev,
			lock_mem_dev,
			grid_response_dev,
			outer_r_dev,
			img_rows,
			img_cols,
			memo_dev
			);
		CUT_CHECK_ERROR("nv_cuda_keypoint_select() failed\n");
		// sync
		CUDA_SAFE_CALL(hipMemcpy(&nkeypoint, nkeypoint_dev, sizeof(int), hipMemcpyDeviceToHost));

		nv_cuda_keypoint_edge_thresh<<<dim3(nkeypoint), dim3(NV_CUDA_KEYPOINT_RADIUS_MAX * 2 + 1)>>>(
			nkeypoint,
			keypoints_tmp_dev,
			outer_r_dev,
			memo_dev);

		tmp = keypoints_tmp->v;
		CUDA_SAFE_CALL(hipMemcpy(keypoints_tmp, keypoints_tmp_dev, 
			sizeof(nv_matrix_t), hipMemcpyDeviceToHost));
		CUDA_SAFE_CALL(hipMemcpy(tmp, keypoints_tmp->v, 
			nkeypoint * keypoints_tmp->step * sizeof(float), hipMemcpyDeviceToHost));
		keypoints_tmp->v = tmp;
		{
			int i, j = 0;
			for (i = 0; i < nkeypoint; ++i) {
				if (fabsf(NV_MAT_V(keypoints_tmp, i, NV_KEYPOINT_RESPONSE_IDX)) > NV_KEYPOINT_THRESH) {
					if (i != j) {
						nv_vector_copy(keypoints_tmp, j, keypoints_tmp, i);
					}
					++j;
				}
			}
			nkeypoint = j;
		}
		qsort(keypoints_tmp->v, nkeypoint, keypoints_tmp->step * sizeof(float), nv_cuda_keypoint_desc_cmp);
		nkeypoint = NV_MIN(keypoints->m, nkeypoint);
		nv_matrix_m(keypoints_tmp, nkeypoint);

		keypoints_dev = nv_cuda_matrix_dup(keypoints_tmp);
#if BENCHMARK
		printf("- select : %ldms\n", nv_clock() - t);
		t = nv_clock();
#endif
	}
	/* orientation */
	{
		dim3 histdata_blocks(NV_MAX(nkeypoint, 1));
		dim3 histdata_threads(NV_KEYPOINT_HIST_SAMPLE, NV_KEYPOINT_HIST_SAMPLE);
		dim3 orientation_blocks(NV_MAX(nkeypoint, 1));
		dim3 orientation_threads(NV_KEYPOINT_HIST_SAMPLE);

		float *tmp;	

		CUDA_SAFE_CALL(hipMalloc(&histdata_dev,
			sizeof(nv_cuda_keypoint_histdata_t) * nkeypoint * NV_KEYPOINT_HIST_SAMPLE * NV_KEYPOINT_HIST_SAMPLE));

		nv_cuda_keypoint_orientation_histdata<<<histdata_blocks, histdata_threads>>>(
			nkeypoint,
			keypoints_dev,
			memo_dev,
			histdata_dev
			);
		CUT_CHECK_ERROR("nv_cuda_keypoint_orientation_histdata() failed\n");
		nv_cuda_keypoint_orientation<<<orientation_blocks, orientation_threads>>>(
			nkeypoint,
			keypoints_dev,
			histdata_dev
			);
		CUT_CHECK_ERROR("nv_cuda_keypoint_orientation() failed\n");
		tmp = keypoints->v;
		CUDA_SAFE_CALL(hipMemcpy(keypoints, keypoints_dev, 
			sizeof(nv_matrix_t), hipMemcpyDeviceToHost));
		CUDA_SAFE_CALL(hipMemcpy(tmp, keypoints->v, 
			nkeypoint * keypoints->step * sizeof(float), hipMemcpyDeviceToHost));
		keypoints->v = tmp;
#if BENCHMARK
		printf("- orientation : %ldms\n", nv_clock() - t);
		t = nv_clock();
#endif
	}
	{
		dim3 histdata_blocks(NV_MAX(nkeypoint, 1), NV_CUDA_KEYPOINT_DESC_M);
		dim3 histdata_threads(NV_KEYPOINT_HIST_SAMPLE, NV_KEYPOINT_HIST_SAMPLE);
		dim3 desc_blocks(NV_MAX(nkeypoint, 1), NV_CUDA_KEYPOINT_DESC_M);
		dim3 desc_threads(NV_KEYPOINT_HIST_SAMPLE);
		float *tmp;

		hipFree(histdata_dev);
		CUDA_SAFE_CALL(hipMalloc(&histdata_dev,
			sizeof(nv_cuda_keypoint_histdata_t) * nkeypoint 
			* NV_KEYPOINT_HIST_SAMPLE * NV_KEYPOINT_HIST_SAMPLE * NV_CUDA_KEYPOINT_DESC_M));
		desc_dev = nv_cuda_matrix_alloc_zero(desc->n, nkeypoint);

		nv_cuda_keypoint_desc_histdata<<<histdata_blocks, histdata_threads>>>(
			nkeypoint,
			keypoints_dev,
			memo_dev,
			histdata_dev
			);
		//CUT_CHECK_ERROR("nv_cuda_keypoint_desc_histdata() failed\n");
		nv_cuda_keypoint_desc<<<desc_blocks, desc_threads>>>(
			nkeypoint,
			desc_dev,
			histdata_dev);
		//CUT_CHECK_ERROR("nv_cuda_keypoint_desc() failed\n");
		tmp = desc->v;
		CUDA_SAFE_CALL(hipMemcpy(desc, desc_dev, 
			sizeof(nv_matrix_t), hipMemcpyDeviceToHost));
		CUDA_SAFE_CALL(hipMemcpy(tmp, desc->v, 
			nkeypoint * desc->step * sizeof(float), hipMemcpyDeviceToHost));
		desc->v = tmp;
#if BENCHMARK
		printf("- desc : %ldms\n", nv_clock() - t);
		t = nv_clock();
#endif
	}
	nv_cuda_matrix_free(inner_r_dev);
	nv_cuda_matrix_free(outer_r_dev);
	nv_cuda_matrix_free(integral_dev);
	nv_cuda_matrix_free(integral_tilted_dev);
	nv_cuda_matrix_free(scale_response_dev);
	nv_cuda_matrix_free(keypoints_tmp_dev);
	nv_cuda_matrix_free(keypoints_dev);
	nv_cuda_matrix_free(grid_response_dev);
	nv_cuda_matrix_free(desc_dev);

	for (i = 0; i < NV_KEYPOINT_LEVEL; ++i) {
		nv_cuda_matrix_free(memo[i]);
	}
	hipFree(memo_dev);
	hipFree(nkeypoint_dev);
	hipFree(lock_mem_dev);
	hipFree(area_inv_table_dev);
	hipFree(area_table_dev);
	hipFree(histdata_dev);

	nv_matrix_free(&keypoints_tmp);
	nv_matrix_free(&outer_r);
	nv_matrix_free(&inner_r);
	nv_matrix_free(&integral);
	nv_matrix_free(&integral_tilted);
	nv_free(memo);

	return nkeypoint;
}