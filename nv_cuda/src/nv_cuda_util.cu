#include "hip/hip_runtime.h"
/*
 * This file is part of libnv.
 *
 * Copyright (C) 2012 nagadomi@nurs.or.jp
 * 
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License,
 * or any later version.
 * 
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <cutil_inline.h>
//#include <cutil.h>
//#include <hipblas.h>
#include "nv_core.h"
#include "nv_cuda_util.h"

static int nv_cuda_is_available = 0;
static int nv_cuda_sm_count = 0;
static int nv_cuda_thread_max = 0;

int 
nv_cuda_init(void)
{
#if __DEVICE_EMULATION__
	nv_cuda_is_available = 1;
	nv_cuda_sm_count = 16;
	nv_cuda_thread_max = NV_CUDA_THREAD_MAX;
	return 0;
#else
	int count = 0;
	int i = 0;
	hipDeviceProp_t prop;
    
	if (hipGetDeviceCount(&count) != hipSuccess) {
		return -1;
	}
	if(count == 0) {
		return -1;
	}
	
	for(i = 0; i < count; i++) {
		if(hipGetDeviceProperties(&prop, i) == hipSuccess) {
			if(prop.major >= 1) {
				nv_cuda_sm_count = prop.multiProcessorCount;
				nv_cuda_thread_max = prop.maxThreadsPerBlock;

				// max
				if (nv_cuda_thread_max > NV_CUDA_THREAD_MAX) {
					nv_cuda_thread_max = NV_CUDA_THREAD_MAX;
				}

				break;
			}
		}
	}
	if(i == count) {
		return -1;
	}
	if (hipSetDevice(i) != hipSuccess) {
		return -1;
	}

	nv_cuda_is_available = 1;

	return 0;
#endif
}


int 
nv_cuda_available(void)
{
	return nv_cuda_is_available;
}


int 
nv_cuda_block(int n)
{
	if (n < nv_cuda_sm_count) {
		return 1;
	}
	if (n < nv_cuda_sm_count * 32) {
		return n / 32 + (n % 32 != 0 ? 1:0);
	}
	if (n < nv_cuda_sm_count * nv_cuda_thread_max) {
		return nv_cuda_sm_count;
	}
	return n / nv_cuda_thread_max + (n % nv_cuda_thread_max != 0 ? 1:0);
}

int 
nv_cuda_thread(int n)
{
	if (n < nv_cuda_sm_count) {
		return n;
	}
	if (n < nv_cuda_sm_count * 32) {
		return 32;
	}

	if (n < nv_cuda_sm_count * nv_cuda_thread_max) {
		return n / nv_cuda_sm_count 
			+ (32 - (nv_cuda_sm_count >= 32 ? 0:n % nv_cuda_sm_count));
	}
	return nv_cuda_thread_max;
}

int 
nv_cuda_optz_block()
{
	return nv_cuda_sm_count;
}

int 
nv_cuda_optz_thread()
{
	return nv_cuda_thread_max > 32 ? 32: nv_cuda_thread_max;
}

nv_matrix_t *
nv_cuda_matrix_dup(const nv_matrix_t *mat)
{
	nv_matrix_t *dev_mat = NULL;
	nv_matrix_t *dup = nv_matrix_clone(mat);

	CUDA_SAFE_CALL(hipMalloc((void **)&dev_mat, sizeof(nv_matrix_t)));
	CUDA_SAFE_CALL(hipMalloc((void **)&dup->v, (size_t)(mat->list * mat->list_step * sizeof(float))));
	CUDA_SAFE_CALL(hipMemcpy(dup->v, mat->v, (size_t)(mat->list * mat->list_step * sizeof(float)), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpy(dev_mat, dup, sizeof(nv_matrix_t), hipMemcpyHostToDevice));
	nv_matrix_free(&dup);

	return dev_mat;
}

nv_matrix_t *
nv_cuda_matrix_clone(const nv_matrix_t *mat)
{
	return nv_cuda_matrix3d_alloc(mat->n, mat->rows, mat->cols);
}


nv_matrix_t *
nv_cuda_matrix_alloc(int n, int m)
{
	nv_matrix_t *mat = nv_matrix_alloc(n, m);
	nv_matrix_t *dev_mat;

	CUDA_SAFE_CALL(hipMalloc((void **)&dev_mat, sizeof(nv_matrix_t)));
	CUDA_SAFE_CALL(hipMalloc((void **)&mat->v, (size_t)(mat->list * mat->list_step * sizeof(float))));
	CUDA_SAFE_CALL(hipMemcpy(dev_mat, mat, sizeof(nv_matrix_t), hipMemcpyHostToDevice));

	nv_matrix_free(&mat);

	return dev_mat;
}

nv_matrix_t *
nv_cuda_matrix_clone_zero(const nv_matrix_t *mat)
{
	return nv_cuda_matrix3d_alloc_zero(mat->n, mat->rows, mat->cols);
}

nv_matrix_t *
nv_cuda_matrix_alloc_zero(int n, int m)
{
	nv_matrix_t *mat = nv_matrix_alloc(n, m);
	nv_matrix_t *dev_mat;

	CUDA_SAFE_CALL(hipMalloc((void **)&dev_mat, sizeof(nv_matrix_t)));
	CUDA_SAFE_CALL(hipMalloc((void **)&mat->v, (size_t)(mat->list * mat->list_step * sizeof(float))));
	CUDA_SAFE_CALL(hipMemset(mat->v, 0, (size_t)(mat->list * mat->list_step * sizeof(float))));
	CUDA_SAFE_CALL(hipMemcpy(dev_mat, mat, sizeof(nv_matrix_t), hipMemcpyHostToDevice));

	nv_matrix_free(&mat);

	return dev_mat;
}

void
nv_cuda_matrix_zero(nv_matrix_t *mat)
{
	nv_matrix_t host_mat;

	CUDA_SAFE_CALL(hipMemcpy(&host_mat, mat, sizeof(nv_matrix_t), hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipMemset(host_mat.v, 0, (size_t)(host_mat.list * host_mat.list_step * sizeof(float))));
}

nv_matrix_t *
nv_cuda_matrix3d_alloc_zero(int n, int rows, int cols)
{
	nv_matrix_t *mat = nv_matrix3d_alloc(n, rows, cols);
	nv_matrix_t *dev_mat;

	CUDA_SAFE_CALL(hipMalloc((void **)&dev_mat, sizeof(nv_matrix_t)));
	CUDA_SAFE_CALL(hipMalloc((void **)&mat->v, (size_t)(mat->list * mat->list_step * sizeof(float))));
	CUDA_SAFE_CALL(hipMemset(mat->v, 0, (size_t)(mat->list * mat->list_step * sizeof(float))));
	CUDA_SAFE_CALL(hipMemcpy(dev_mat, mat, sizeof(nv_matrix_t), hipMemcpyHostToDevice));

	nv_matrix_free(&mat);

	return dev_mat;
}

nv_matrix_t *
nv_cuda_matrix3d_alloc(int n, int rows, int cols)
{
	nv_matrix_t *mat = nv_matrix3d_alloc(n, rows, cols);
	nv_matrix_t *dev_mat;

	CUDA_SAFE_CALL(hipMalloc((void **)&dev_mat, sizeof(nv_matrix_t)));
	CUDA_SAFE_CALL(hipMalloc((void **)&mat->v, (size_t)(mat->list * mat->list_step * sizeof(float))));
	CUDA_SAFE_CALL(hipMemcpy(dev_mat, mat, sizeof(nv_matrix_t), hipMemcpyHostToDevice));

	nv_matrix_free(&mat);

	return dev_mat;
}

void 
nv_cuda_matrix_free(nv_matrix_t *dev_mat)
{
	nv_matrix_t mat;

	CUDA_SAFE_CALL(hipMemcpy(&mat, dev_mat, sizeof(nv_matrix_t), hipMemcpyDeviceToHost));
	hipFree(mat.v);
	hipFree(dev_mat);
}
